#include "hip/hip_runtime.h"
#include "simulation_acc.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include "globals.hpp"

__global__ void applyGravityKernel(float* forces, float* masses, float gravity, int count);
__global__ void resolveImpulseKernel(float* positions, float* velocities, float* masses, int* radii, int count);
__global__ void applyFrictionKernel(float* velocities, float* positions, int count);
__global__ void integrateKernel(float* positions, float* velocities, float* forces, float* masses, float dt, int count);
__global__ void wallCollisionKernel(int height, int width, float* positions, float* velocities, float* forces, float* masses, int* radii, float dt, int count);
__global__ void clearForcesKernel(float* forces, int count);

SimulationAcc::SimulationAcc(int width, int height, float gravity, int cellSize)
    : width(width), height(height), gravity(gravity), cellSize(cellSize),
        d_positions(nullptr), d_velocities(nullptr), d_forces(nullptr),
        d_masses(nullptr), d_radii(nullptr), num_particles(0){}

SimulationAcc::~SimulationAcc() {
    freeDeviceMemory();
}

void SimulationAcc::allocateDeviceMemory(int count){
    num_particles = count;

    hipMalloc(&d_positions, 2*sizeof(float)*count);
    hipMalloc(&d_velocities, 2*sizeof(float)*count);
    hipMalloc(&d_forces, 2*sizeof(float)*count);
    hipMalloc(&d_masses, 2*sizeof(float)*count);
    hipMalloc(&d_radii, 2*sizeof(int)*count);
}

void SimulationAcc::freeDeviceMemory() {
    hipFree(d_positions);
    hipFree(d_velocities);
    hipFree(d_forces);
    hipFree(d_masses);
    hipFree(d_radii);

    d_positions = d_velocities = d_forces = d_masses = nullptr;
    d_radii = nullptr;
    num_particles = 0;
}

void SimulationAcc::uploadParticles(const std::vector<Particle>& particles){
    int n = particles.size();

    if(n!=num_particles){
        freeDeviceMemory();
        allocateDeviceMemory(n);
    }

    std::vector<float> h_positions(2*n);
    std::vector<float> h_velocities(2*n);
    std::vector<float> h_forces(2*n);
    std::vector<float> h_masses(2*n);
    std::vector<int> h_radii(2*n);

    for(int i=0; i< n; ++i){
        h_positions[2*i] = particles[i].getX();
        h_positions[2*i+1] = particles[i].getY();

        h_velocities[2*i] = particles[i].getVX();
        h_velocities[2*i+1] = particles[i].getVY();

        h_masses[i] = particles[i].getMass();
        h_radii[i] = particles[i].getRadius();
    }

    hipMemcpy(d_positions, h_positions.data(), sizeof(float)*2*n, hipMemcpyHostToDevice);
    hipMemcpy(d_velocities, h_velocities.data(), sizeof(float)*2*n, hipMemcpyHostToDevice);
    hipMemcpy(d_forces, h_forces.data(), sizeof(float)*2*n, hipMemcpyHostToDevice);
    hipMemcpy(d_masses, h_masses.data(), sizeof(float)*2*n, hipMemcpyHostToDevice);
    hipMemcpy(d_masses, h_masses.data(), sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(d_radii, h_radii.data(), sizeof(int)*n, hipMemcpyHostToDevice);

}

void SimulationAcc::downloadParticles(std::vector<Particle>& particles){
    int n = particles.size();

    std::vector<float> h_positions(2*n);
    std::vector<float> h_velocities(2*n);

    hipMemcpy(h_positions.data(), d_positions, sizeof(float)*2*n, hipMemcpyDeviceToHost);
    hipMemcpy(h_velocities.data(), d_velocities, sizeof(float)*2*n, hipMemcpyDeviceToHost);

    for(int i=0;i<n;++i){
        particles[i].setPosition(h_positions[2*i], h_positions[2*i+1]);
        particles[i].setVelocity(h_velocities[2*i], h_velocities[2*i+1]);
    }
}

void SimulationAcc::step(std::vector<Particle>& particles, float dt) {
    
    if (particles.empty()) return;

    memoryTransferInProgress = true;

    // 1. Upload data to GPU
    std::cout<<"Uploading Particles...\n";
    uploadParticles(particles);


    // 2. Compute physics using CUDA kernel(s)
    std::cout<<"Doing the Math...\n";
    computePhysicsGPU(dt);

    // 3. Download updated data back to CPU
    downloadParticles(particles);

    
    memoryTransferInProgress = false;
}

void SimulationAcc::computePhysicsGPU(float dt) {
    int threadsPerBlock = 256;
    int blocks = (num_particles + threadsPerBlock - 1) / threadsPerBlock;

    clearForcesKernel<<<blocks, threadsPerBlock>>>(d_forces, num_particles);

    // 1. Apply gravity
    applyGravityKernel<<<blocks, threadsPerBlock>>>(d_forces, d_masses, gravity, num_particles);

    // 2. Resolve impulse (naive n^2 for now, will optimize later)
    resolveImpulseKernel<<<blocks, threadsPerBlock>>>(d_positions, d_velocities, d_masses, d_radii, num_particles);

    // 3. Apply friction
    // applyFrictionKernel<<<blocks, threadsPerBlock>>>(d_velocities, d_positions, num_particles);

    // 4. Integrate motion
    integrateKernel<<<blocks, threadsPerBlock>>>(
        d_positions, d_velocities, d_forces, d_masses, dt, num_particles
    );

    wallCollisionKernel<<<blocks, threadsPerBlock>>>(height, width, d_positions, d_velocities, d_forces, d_masses, d_radii, dt, num_particles);

    hipDeviceSynchronize(); // Optional: For error debugging
}

__global__ void applyGravityKernel(float* forces, float* masses, float gravity, int count){
    int idx = blockIdx.x* blockDim.x + threadIdx.x;

    if(idx >= count) return;

    forces[2*idx] += 0.0f;
    forces[2*idx+1] += masses[idx]*gravity;
}

__global__ void resolveImpulseKernel(float* positions, float* velocities, float* masses, int* radii, int count){
    int i = blockIdx.x* blockDim.x + threadIdx.x;
    if (i>=count) return;

    float xi = positions[2*i];
    float yi = positions[2*i+1];
    float vxi = velocities[2*i];
    float vyi = velocities[2*i+1];
    float mi = masses[i];
    int ri = radii[i];

    for(int j=0; j<count;++j){
        if(i==j) continue;

        float xj = positions[2*j];
        float yj = positions[2*j+1];

        float dx = xj - xi;
        float dy = yj - yi;

        float dist2 = dx*dx + dy*dy;
        float rj = radii[j];
        float minDist = ri+rj;

        if(dist2 < minDist*minDist){
            // normalize direction
            float dist = sqrtf(dist2);
            if (dist < 1e-6f) continue;

            float nx = dx/dist;
            float ny = dy/dist;

            float penetration = minDist - dist;

            float rvx = velocities[2*j] - velocities[2*i];
            float rvy = velocities[2*j+1] - velocities[2*i+1]; 

            float velAlongNormal = rvx * nx + rvy * ny;
            if (velAlongNormal > 0) continue;

            float e = 0.3f;
            float m1 = masses[i];
            float m2 = masses[j];

            float impulse = -(1.0f + e) * velAlongNormal / (1.0f / m1 + 1.0f / m2);

            float impulseX = impulse * nx * 0.8;
            float impulseY = impulse * ny * 0.8;

            // Apply only to i for stability
            velocities[2*i] -= impulseX / m1;
            velocities[2*i+1] -= impulseY / m1;

            // Soft positional correction
            float percent = 0.2f;
            float slop = 0.01f;
            float correctionMag = fmaxf(penetration - slop, 0.0f) / (1.0f / m1 + 1.0f / m2) * percent;

            positions[2*i] -= correctionMag * nx / m1;
            positions[2*i+1] -= correctionMag * ny / m1;


        }
    }
}

__global__ void applyFrictionKernel(float* velocities, float* positions, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;

    float friction_coefficient = 0.3f; // Tunable constant
    velocities[2 * idx] *= friction_coefficient;
    velocities[2 * idx + 1] *= friction_coefficient;
}

__global__ void integrateKernel(float* positions, float* velocities, float* forces, float* masses, float dt, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;

    float ax = forces[2 * idx] / masses[idx];
    float ay = forces[2 * idx + 1] / masses[idx];

    velocities[2 * idx] += ax * dt;
    velocities[2 * idx + 1] += ay * dt;

    positions[2 * idx] += velocities[2 * idx] * dt;
    positions[2 * idx + 1] += velocities[2 * idx + 1] * dt;
   
}

__global__ void wallCollisionKernel(int height, int width, float* positions, float* velocities, float* forces, float* masses, int* radii,float dt, int count){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;

    float radius = radii[idx];

    float x = positions[idx*2];
    float y = positions[idx*2+1];

    if(x - radius < 0){
        velocities[idx*2] = - velocities[idx*2]*0.8f;
        positions[idx*2] = radius;
    }
    if(x+radius > width){
        velocities[idx*2] = - velocities[idx*2]*0.8f;
        positions[idx*2] = width - radius;
    }
    if(y - radius < 0){
        velocities[idx*2+1] = -velocities[idx*2+1]*0.8f;
        positions[idx*2+1] = radius;
    }
    if(y+radius > height){
        velocities[idx*2+1] = -velocities[idx*2+1]*0.8f;
        positions[idx*2+1] = height - radius;
    }
}

__global__ void clearForcesKernel(float* forces, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;
    forces[2 * idx] = 0.0f;
    forces[2 * idx + 1] = 0.0f;
}
