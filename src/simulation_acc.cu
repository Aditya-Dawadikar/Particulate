#include "hip/hip_runtime.h"
#include "simulation_acc.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include "globals.hpp"

__global__ void applyGravityKernel(float* forces, float* masses, float gravity, int count);
__global__ void resolveImpulseKernel(float* forces, float* positions, float* velocities, float* masses, int* radii, int count, float dt);
__global__ void applyFrictionKernel(float* velocities, float* positions, int count);
__global__ void integrateKernel(float* positions, float* velocities, float* forces, float* masses, float dt, int count);
__global__ void wallCollisionKernel(int height, int width, float* positions, float* velocities, float* forces, float* masses, int* radii, float dt, int count);
__global__ void clearForcesKernel(float* forces, int count);

SimulationAcc::SimulationAcc(int width, int height, int cellSize)
    : width(width), height(height), gravity(980.0f), cellSize(cellSize),
        d_positions(nullptr), d_velocities(nullptr), d_forces(nullptr),
        d_masses(nullptr), d_radii(nullptr), num_particles(0){}

SimulationAcc::~SimulationAcc() {
    freeDeviceMemory();
}

void SimulationAcc::allocateDeviceMemory(int count){
    num_particles = count;

    hipMalloc(&d_positions, 2*sizeof(float)*count);
    hipMalloc(&d_velocities, 2*sizeof(float)*count);
    hipMalloc(&d_forces, 2*sizeof(float)*count);
    hipMalloc(&d_masses, 2*sizeof(float)*count);
    hipMalloc(&d_radii, 2*sizeof(int)*count);
}

void SimulationAcc::freeDeviceMemory() {
    hipFree(d_positions);
    hipFree(d_velocities);
    hipFree(d_forces);
    hipFree(d_masses);
    hipFree(d_radii);

    d_positions = d_velocities = d_forces = d_masses = nullptr;
    d_radii = nullptr;
    num_particles = 0;
}

void SimulationAcc::uploadParticles(const std::vector<Particle>& particles){
    int n = particles.size();

    if(n!=num_particles){
        freeDeviceMemory();
        allocateDeviceMemory(n);
    }

    std::vector<float> h_positions(2*n);
    std::vector<float> h_velocities(2*n);
    std::vector<float> h_forces(2*n);
    std::vector<float> h_masses(n);
    std::vector<int> h_radii(n);

    for(int i=0; i< n; ++i){
        h_positions[2*i] = particles[i].getX();
        h_positions[2*i+1] = particles[i].getY();

        h_velocities[2*i] = particles[i].getVX();
        h_velocities[2*i+1] = particles[i].getVY();

        h_masses[i] = particles[i].getMass();
        h_radii[i] = particles[i].getRadius();
    }

    hipMemcpy(d_positions, h_positions.data(), sizeof(float)*2*n, hipMemcpyHostToDevice);
    hipMemcpy(d_velocities, h_velocities.data(), sizeof(float)*2*n, hipMemcpyHostToDevice);
    hipMemcpy(d_forces, h_forces.data(), sizeof(float)*2*n, hipMemcpyHostToDevice);
    hipMemcpy(d_masses, h_masses.data(), sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(d_radii, h_radii.data(), sizeof(int)*n, hipMemcpyHostToDevice);

}

void SimulationAcc::downloadParticles(std::vector<Particle>& particles){
    int n = particles.size();

    std::vector<float> h_positions(2*n);
    std::vector<float> h_velocities(2*n);

    hipMemcpy(h_positions.data(), d_positions, sizeof(float)*2*n, hipMemcpyDeviceToHost);
    hipMemcpy(h_velocities.data(), d_velocities, sizeof(float)*2*n, hipMemcpyDeviceToHost);

    for(int i=0;i<n;++i){
        particles[i].setPosition(h_positions[2*i], h_positions[2*i+1]);
        particles[i].setVelocity(h_velocities[2*i], h_velocities[2*i+1]);
    }
}

void SimulationAcc::step(std::vector<Particle>& particles, float dt) {
    
    if (particles.empty()) return;

    memoryTransferInProgress = true;

    // 1. Upload data to GPU
    std::cout<<"Uploading Particles...\n";
    uploadParticles(particles);


    // 2. Compute physics using CUDA kernel(s)
    std::cout<<"Doing the Math...\n";
    // computePhysicsGPU(dt);
    int num_substeps = 4;
    float substep_dt = dt / num_substeps;

    // Run simulation in substeps
    for (int i = 0; i < num_substeps; ++i) {
        computePhysicsGPU(substep_dt);
    }


    // 3. Download updated data back to CPU
    downloadParticles(particles);

    
    memoryTransferInProgress = false;
}

void SimulationAcc::computePhysicsGPU(float dt) {
    int threadsPerBlock = 256;
    int blocks = (num_particles + threadsPerBlock - 1) / threadsPerBlock;

    clearForcesKernel<<<blocks, threadsPerBlock>>>(d_forces, num_particles);

    // 1. Apply gravity
    applyGravityKernel<<<blocks, threadsPerBlock>>>(d_forces, d_masses, gravity, num_particles);

    // 2. Resolve impulse (naive n^2 for now, will optimize later)
    resolveImpulseKernel<<<blocks, threadsPerBlock>>>(d_forces, d_positions, d_velocities, d_masses, d_radii, num_particles, dt);

    // 3. Apply friction
    // applyFrictionKernel<<<blocks, threadsPerBlock>>>(d_velocities, d_positions, num_particles);

    // 4. Integrate motion
    integrateKernel<<<blocks, threadsPerBlock>>>(
        d_positions, d_velocities, d_forces, d_masses, dt, num_particles
    );

    wallCollisionKernel<<<blocks, threadsPerBlock>>>(height, width, d_positions, d_velocities, d_forces, d_masses, d_radii, dt, num_particles);

    hipDeviceSynchronize(); // Optional: For error debugging
}

__global__ void applyGravityKernel(float* forces, float* masses, float gravity, int count){
    int idx = blockIdx.x* blockDim.x + threadIdx.x;

    if(idx >= count) return;

    forces[2*idx] += 0.0f;
    forces[2*idx+1] += masses[idx]*gravity;
}

__global__ void resolveImpulseKernel(float* forces, float* positions, float* velocities, float* masses, int* radii, int count, float dt){
    int i = blockIdx.x* blockDim.x + threadIdx.x;
    if (i>=count) return;

    float xi = positions[2*i];
    float yi = positions[2*i+1];
    float vxi = velocities[2*i];
    float vyi = velocities[2*i+1];
    float mi = masses[i];
    int ri = radii[i];

    for(int j=0; j<count;++j){
        if(i==j) continue;

        float xj = positions[2*j];
        float yj = positions[2*j+1];
        float vxj = velocities[2*j];
        float vyj = velocities[2*j+1];
        float mj = masses[j];
        int rj = radii[j];

        float dx = xj - xi;
        float dy = yj - yi;
        float dist2 = dx*dx + dy*dy;
        float minDist = ri + rj;

        if(dist2 < minDist*minDist && dist2 > 0.0001f){
            float dist = sqrtf(dist2);
            float nx = dx / dist;
            float ny = dy / dist;

            // Relative velocity
            float rvx = vxj - vxi;
            float rvy = vyj - vyi;

            // Velocity along normal
            float velAlongNormal = rvx * nx + rvy * ny;

            // Only resolve if particles are moving toward each other
            if (velAlongNormal > 0) continue;

            // Coefficient of restitution (0 = perfectly inelastic, 1 = perfectly elastic)
            float e = 0.2f;  // Adjust as needed

            // Calculate impulse scalar
            float impulse = -(1.0f + e) * velAlongNormal;
            impulse /= (1.0f/mi + 1.0f/mj);

            // Apply impulse to both particles (equal and opposite)
            float impulseX = impulse * nx;
            float impulseY = impulse * ny;

            // Update velocities (atomic operations for thread safety)
            atomicAdd(&velocities[2*i], -impulseX / mi);
            atomicAdd(&velocities[2*i+1], -impulseY / mi);
            atomicAdd(&velocities[2*j], impulseX / mj);
            atomicAdd(&velocities[2*j+1], impulseY / mj);

            // Positional correction to prevent sinking
            float percent = 0.2f; // Usually 20% to 80%
            float slop = 0.01f;   // Usually 0.01 to 0.1
            float penetration = minDist - dist;
            penetration=(penetration, 2.0f);
            float correction = fmaxf(penetration - slop, 0.0f) / (1.0f/mi + 1.0f/mj) * percent;

            float correctionX = correction * nx;
            float correctionY = correction * ny;

            // Apply positional correction (atomic operations for thread safety)
            atomicAdd(&positions[2*i], -correctionX / mi);
            atomicAdd(&positions[2*i+1], -correctionY / mi);
            atomicAdd(&positions[2*j], correctionX / mj);
            atomicAdd(&positions[2*j+1], correctionY / mj);

        }
    }
}

__global__ void applyFrictionKernel(float* velocities, float* positions, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;

    float friction_coefficient = 0.3f; // Tunable constant
    velocities[2 * idx] *= friction_coefficient;
    velocities[2 * idx + 1] *= friction_coefficient;
}

__global__ void integrateKernel(float* positions, float* velocities, float* forces, float* masses, float dt, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;

    float ax = forces[2 * idx] / masses[idx];
    float ay = forces[2 * idx + 1] / masses[idx];

    velocities[2 * idx] += ax * dt;
    velocities[2 * idx + 1] += ay * dt;

    positions[2 * idx] += velocities[2 * idx] * dt;
    positions[2 * idx + 1] += velocities[2 * idx + 1] * dt;
   
}

__global__ void wallCollisionKernel(int height, int width, float* positions, float* velocities, float* forces, float* masses, int* radii,float dt, int count){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;

    float radius = radii[idx];

    float x = positions[idx*2];
    float y = positions[idx*2+1];

    if(x - radius < 0){
        velocities[idx*2] = - velocities[idx*2]*0.8f;
        positions[idx*2] = radius;
    }
    if(x+radius > width){
        velocities[idx*2] = - velocities[idx*2]*0.8f;
        positions[idx*2] = width - radius;
    }
    if(y - radius < 0){
        velocities[idx*2+1] = -velocities[idx*2+1]*0.8f;
        positions[idx*2+1] = radius;
    }
    if(y+radius > height){
        velocities[idx*2+1] = -velocities[idx*2+1]*0.8f;
        positions[idx*2+1] = height - radius;
    }
}

__global__ void clearForcesKernel(float* forces, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count) return;
    forces[2 * idx] = 0.0f;
    forces[2 * idx + 1] = 0.0f;
}
